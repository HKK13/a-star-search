#include <queue>;
#include <unordered_map>
#include <iostream>;
#include <string>;
#include <limits>;
#include <cstdio>
#include <ctime>
#include <vector>;
#include "hip/hip_runtime.h"


#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

// Dummy kernel
__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

struct SimpleGrid {
  int ** grid;
  int width, height;

  SimpleGrid(int ** grid, int width, int height) {
    this->grid = grid;
    this->width = width;
    this->height = height;
  }

  int GetCost(std::pair<int, int> from, std::pair<int, int> to) {
    return 1;
  }

  int * operator [](int i) const { return this->grid[i]; }
};


struct pair_hash {
  std::size_t operator () (const std::pair<int, int> &p) const {
    size_t h = (p.first * (std::numeric_limits<int>::max() + 1) + p.second);
    return h;
  }
};


std::unordered_map<std::pair<int, int>, std::pair<int, int>, pair_hash>
astar(SimpleGrid grid, std::pair<int, int> start, std::pair<int, int> goal) {

  // Item to be stored in PQ
  typedef std::pair<int, std::pair<int, int>> PQElement;
  // PQ definition
  std::priority_queue<PQElement, std::vector<PQElement>, std::less<PQElement>> frontier;

  // Insert starting position
  frontier.emplace(0, start);

  // Utility memory lists
  std::unordered_map<std::pair<int, int>, int, pair_hash> costSoFar;
  std::unordered_map<std::pair<int, int>, std::pair<int, int>, pair_hash> cameFrom;

  cameFrom[start] = start;
  costSoFar[start] = 0;

  // Begin searching until the goal is reached or every possible value
  // evaluated with a failure.
  while (!frontier.empty()) {

    // Get a node to expand it.
    int x, y;
    std::tie(x, y) = frontier.top().second;
    frontier.pop();

    // If goal terminate.
    if (x == goal.first && y == goal.second) break;

#ifdef _DEBUG
    std::cout << "Visiting " << x << " " << y << std::endl;
#endif // DEBUG

    // Add neighbors to priority queue if they are passable.
    for (int i = -1; i < 2; i++) {
      for (int j = -1; j < 2; j++) {

        // Ignore itself.
        if ((i == 0) && (j == 0)) { continue; }

        // Check if it is passable and a valid point in the grid.
        if (((y + i >= 0) && (y + i < grid.height))
          && ((x + j) >= 0) && (x + j < grid.width)
          && grid[y + i][x + j] != 0) {

          std::pair<int, int> next = std::make_pair(x + j, y + i);
          int newCost = costSoFar[std::make_pair(x, y)] + 1;

          // If it the position is explored or current is cheaper than existing.
          if ((costSoFar.find(next) == costSoFar.end()) || (newCost < costSoFar[next])) {
            int priority = newCost + (std::abs(x - next.first) + std::abs(y - next.second));
            frontier.emplace(priority, next);
            cameFrom[next] = std::make_pair(x, y);
            costSoFar[next] = newCost;
          }
        }
      }
    }
  }

  // Free memory.
  std::priority_queue<PQElement, std::vector<PQElement>, std::less<PQElement>>().swap(frontier);
  std::unordered_map<std::pair<int, int>, int, pair_hash>().swap(costSoFar);

  return cameFrom;
}


void drawGrid(SimpleGrid grid, std::pair<int, int> goal,
  std::unordered_map<std::pair<int, int>, std::pair<int, int>, pair_hash> parents) {
  std::pair<int, int> current = goal;
  do {
    if (grid[current.second][current.first] != 0) {
      grid[current.second][current.first] = 6;
    }
    else {
      grid[current.second][current.first] = 99;
    }

    if (current == parents[current]) break;

    current = parents[current];
  } while (parents.find(goal) != parents.end());

  for (int i = 0; i < grid.height; i++) {
    for (int j = 0; j < grid.width; j++) {
      std::cout << grid[i][j] << " ";
    }
    std::cout << std::endl;
  }
}


int main() {
  int constantGrid[][10] = {
    { 1, 0, 1, 1, 1, 1, 1, 1, 1, 1 },
    { 1, 0, 1, 1, 1, 1, 1, 1, 1, 1 },
    { 1, 0, 1, 1, 1, 1, 1, 1, 1, 1 },
    { 1, 0, 1, 0, 1, 0, 0, 1, 1, 1 },
    { 1, 0, 1, 0, 1, 0, 0, 1, 1, 1 },
    { 1, 1, 1, 0, 1, 0, 0, 1, 1, 1 },
    { 1, 1, 1, 0, 1, 1, 1, 1, 1, 1 },
    { 1, 1, 1, 0, 1, 1, 0, 0, 1, 1 },
    { 1, 1, 1, 0, 1, 1, 0, 0, 1, 1 },
    { 1, 1, 1, 0, 1, 1, 1, 1, 1, 1 }
  };


  int ** grid = new int*[10];
  for (int i = 0; i < 10; i++) {
    grid[i] = new int[10];
    for (int j = 0; j < 10; j++) {
      grid[i][j] = constantGrid[i][j];
    }
  }

  SimpleGrid simpleGrid = SimpleGrid(grid, 10, 10);

  std::clock_t start;
  double duration;

  start = std::clock();

  std::unordered_map<std::pair<int, int>, std::pair<int, int>, pair_hash> parents;
  parents = astar(simpleGrid, std::make_pair(0, 0), std::make_pair(9, 9));

  duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;

  std::cout << "Duration: " << duration << " seconds\n";

  drawGrid(simpleGrid, std::make_pair(9, 9), parents);

  std::unordered_map<std::pair<int, int>, std::pair<int, int>, pair_hash>().swap(parents);

  return 0;
}